#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""
#include "timemeasure.h"

#include <stdio.h>

__global__ void addKernel(size_t n)
{
    int c = 0;
    for (size_t i = 0; i < n; i++)
    {
        c++;
    }
}

TimeMeasure tm{};
TimeMeasure tm2{};

void DisplayHeader()
{
    const int kb = 1024;
    const int mb = kb * kb;

    int devCount;
    hipGetDeviceCount(&devCount);
    std::cout << "CUDA Devices: " << std::endl << std::endl;


    for (int dev = 0; dev < devCount; ++dev) {
        hipSetDevice(dev);
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

        printf("Mrmp: %i\n", deviceProp.regsPerMultiprocessor);
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCudaStreams(size_t n, int times)
{
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    hipStream_t* streams = (hipStream_t*)malloc(sizeof(hipStream_t) * times);
    for (size_t i = 0; i < times; i++)
    {
        hipStreamCreateWithFlags(&streams[i], hipStreamNonBlocking);
    }

    // Launch a kernel on the GPU with one thread for each element.
    for (int i = 0; i < times; i++) {
        tm2.StartCounter();
        addKernel << <1, 1, 0, streams[i] >> > (n);
        //cudaStatus = hipDeviceSynchronize();
        double time_ms = tm2.GetCounter();
        printf("Process %i launched. t: %f Seconds\n", i, time_ms / 1000);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    hipFree(dev_c);
    free(streams);

    return cudaStatus;
}


// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCudaSec(size_t n, int times)
{
    int* dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    for (int i = 0; i < times; i++) {
        tm2.StartCounter();
        addKernel << <1, 1>> > (n);
        //cudaStatus = hipDeviceSynchronize();
        double time_ms = tm2.GetCounter();
        printf("Process %i launched. t: %f Seconds\n", i, time_ms / 1000);
    }

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

Error:
    hipFree(dev_c);

    return cudaStatus;
}

int main(int argc, char *argv[])
{
    if (argc > 1 && argc < 3) {
        printf("Error, insuficientes argumentos\nUso: %s iteraciones total_kernels\n", argv[0]);
        exit(1);
    }

    DisplayHeader();

    int iters = 20000000;
    int times = 5;

    if (argc > 1) {
        sprintf(argv[1], "%i", &iters);
        sprintf(argv[2], "%i", &times);
    }

    // Add vectors in parallel.
    tm.StartCounter();
    hipError_t cudaStatus = addWithCudaSec(iters, times);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }
    double time_ms = tm.GetCounter();
    printf("All processes finished. t: %f Seconds\n", time_ms/1000);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    printf("He terminado!\n");

    return 0;
}
