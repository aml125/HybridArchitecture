#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <cstdlib>
#include "timemeasure.h"
#include "log.h"

#define THREADED false

typedef struct {
    double x, y;
} Vec2;

__device__ double rosenbrock(Vec2 X, double a, double b) {
    return pow(a - X.x, (double)2) + b * pow(X.y - pow(X.x, (double)2), (double)2);
}

__device__ Vec2 gradient(Vec2 X, double a, double b) {
    Vec2 result = {
        -2 * a + 4 * b * pow(X.x, (double)3) - 4 * b * X.x * X.y + 2 * X.x,
        2 * b * (X.y - pow(X.x, (double)2))
    };
    return result;
}

__device__ Vec2 gradient_descent(Vec2 X, double a, double b, int iters, double step, int i) {
    double eval = rosenbrock(X, a, b);
    //printf("{%i} Initial Eval: %lf\n", i/2, eval);

    for (size_t i = 0; i < iters; i++)
    {
        Vec2 grad = gradient(X, a, b);
        X.x -= step * grad.x;
        X.y -= step * grad.y;
        eval = rosenbrock(X, a, b);
    }
    //printf("{%i} Eval: %lf\n", i/2, eval);
    return X;
}

__global__ void gradientKernel(double* vars, double* res, int iters)
{
    int x = threadIdx.x;
    int y = blockIdx.x;
    int i = (y * blockDim.x + x) * 2;
    Vec2 X = { vars[i], vars[i + 1] };
    gradient_descent(X, 1, 100, iters, 0.002, i);
    //printf("I'm {%i}. X: %lf, Y: %lf\n", i, X.x, X.y);
}


/*
*====================================================================================
*   HOST CODE
*
*====================================================================================
*/

std::atomic_int32_t threadCounter;
std::mutex mtx;

// Helper function for using CUDA to add vectors in parallel.
hipError_t gradientDescentWithCuda(double* vars, double* res, int NPOINTS, int ITERATIONS, double* dev_vars, hipStream_t *stream)
{
    double* dev_res = 0;

    //std::cout << "e" << std::endl;
    auto cudaStatus = hipMalloc((void**)&dev_res, NPOINTS * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //std::cout << "holi" << std::endl;
    gradientKernel << <NPOINTS / 100, 100, 1, *stream >> > (dev_vars, dev_res, ITERATIONS);
    //std::cout << "hola" << std::endl;

    //mtx.lock();
    //std::cout << "lanzado " << std::endl;
    //mtx.unlock();

    /*cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }*/

    

    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}
    //
    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //    goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(res, dev_res, NPOINTS * sizeof(int), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}
//
Error:
    /*mtx.lock();
    int i = ++threadCounter;
    std::cout << "Terminado: " << i << std::endl;
    mtx.unlock();*/
    //hipFree(dev_res);
    return cudaStatus;
}

hipError_t gradientDescentWithCudaThreaded(double* vars, double* res, int NPOINTS, int ITERATIONS, double* dev_vars, hipStream_t* stream)
{
    double* dev_res = 0;

    //std::cout << "e" << std::endl;
    auto cudaStatus = hipMalloc((void**)&dev_res, NPOINTS * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    //std::cout << "holi" << std::endl;
    gradientKernel << <NPOINTS / 100, 100, 1, *stream >> > (dev_vars, dev_res, ITERATIONS);
    //std::cout << "hola" << std::endl;

    mtx.lock();
    std::cout << "lanzado " << std::endl;
    mtx.unlock();

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }



    //// Check for any errors launching the kernel
    //cudaStatus = hipGetLastError();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    //    goto Error;
    //}
    //
    //// hipDeviceSynchronize waits for the kernel to finish, and returns
    //// any errors encountered during the launch.
    //cudaStatus = hipDeviceSynchronize();
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
    //    goto Error;
    //}

    //// Copy output vector from GPU buffer to host memory.
    //cudaStatus = hipMemcpy(res, dev_res, NPOINTS * sizeof(int), hipMemcpyDeviceToHost);
    //if (cudaStatus != hipSuccess) {
    //    fprintf(stderr, "hipMemcpy failed!");
    //    goto Error;
    //}
//
Error:
    mtx.lock();
    int i = ++threadCounter;
    std::cout << "Terminado: " << i << std::endl;
    mtx.unlock();
    //hipFree(dev_res);
    return cudaStatus;
}

void launchKernelOnThread(double* vars, double* res, int NPOINTS, int ITERATIONS, double* dev_vars, hipStream_t* stream) {
    std::thread thread(gradientDescentWithCudaThreaded, vars, res, NPOINTS, ITERATIONS, dev_vars, stream);
    thread.detach();
}

void launchKernel(double* vars, double* res, int NPOINTS, int ITERATIONS, double* dev_vars, hipStream_t* stream, int i) {
    hipError_t cudaStatus = gradientDescentWithCuda(vars, res, NPOINTS, ITERATIONS, dev_vars, stream);
    std::cout << "lanzado: " << i <<  std::endl;
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "gradientKernel failed!");
        exit(-1);
    }
}

int launchTest(int NPOINTS, int ITERATIONS, int processes, hipStream_t *streams) {
    double* vars = new double[NPOINTS * 2];
    double* res = new double[NPOINTS];


    for (size_t i = 0; i < NPOINTS * 2; i++)
    {
        vars[i] = (double)(rand() % 100) / (double)100;
    }

    double* dev_vars = 0;

    hipError_t cudaStatus;
    //std::cout << "a" << std::endl;
    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_vars, NPOINTS * 2 * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        return -1;
    }
    //std::cout << "i" << std::endl;
    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_vars, vars, NPOINTS * 2 * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        return -1;
    }

    threadCounter = 0;

    // Add vectors in parallel.
    for (size_t i = 0; i < processes; i++)
    {
        if (THREADED) {
            launchKernelOnThread(vars, res, NPOINTS, ITERATIONS, dev_vars, &streams[i]);
        }
        else {
            launchKernel(vars, res, NPOINTS, ITERATIONS, dev_vars, &streams[i], i);
        }
    }

    if (THREADED) {
        mtx.lock();
        std::cout << "fin" << std::endl;
        mtx.unlock();

        while (threadCounter < processes)
            Sleep(5);
    }
    else {
        std::cout << "fin" << std::endl;
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
            return -1;
        }
    }

    

    std::cout << "sincronizado" << std::endl;

    hipFree(dev_vars);
    

    return 0;
}

#define MAX_PROCESSES 10

int main()
{
    // Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        return -1;
    }

    TimeMeasure tm;
    TimeMeasure tm2;
    int NPOINTS = 1000;
    int ITERATIONS = 10000;

    hipStream_t streams[MAX_PROCESSES];
    for (size_t i = 0; i < MAX_PROCESSES; i++)
    {
        hipStreamCreate(&streams[i]);
    }

    for (size_t i = 0; i <= MAX_PROCESSES; i+=1)
    {
        tm.StartCounter();
        if (launchTest(NPOINTS, ITERATIONS, i, streams) != 0) {
            std::cout << "ERROR" << std::endl;
            return -1;
        }
        double time = tm.GetCounter();

        GM::log("Proceses: " + std::to_string(i) + " Time: " + std::to_string(time));
    }

    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }
    

    GM::flush();



    return 0;
}


